//
// parrot_optimizer_animated_initialize.cu
//
// Code generation for function 'parrot_optimizer_animated_initialize'
//

// Include files
#include "parrot_optimizer_animated_initialize.h"
#include "_coder_parrot_optimizer_animated_mex.h"
#include "parrot_optimizer_animated_data.h"
#include "rt_nonfinite.h"

// Function Definitions
void parrot_optimizer_animated_initialize()
{
  mex_InitInfAndNan();
  emlrtInitGPU(emlrtRootTLSGlobal);
  hipGetLastError();
  mexFunctionCreateRootTLS();
  emlrtClearAllocCountR2012b(emlrtRootTLSGlobal, false, 0U, nullptr);
  emlrtEnterRtStackR2012b(emlrtRootTLSGlobal);
  emlrtLicenseCheckR2022a(emlrtRootTLSGlobal,
                          "EMLRT:runTime:MexFunctionNeedsLicense",
                          "distrib_computing_toolbox", 2);
  emlrtLicenseCheckR2022a(emlrtRootTLSGlobal,
                          "EMLRT:runTime:MexFunctionNeedsLicense",
                          "statistics_toolbox", 2);
  emlrtFirstTimeR2012b(emlrtRootTLSGlobal);
}

// End of code generation (parrot_optimizer_animated_initialize.cu)
