#include "hip/hip_runtime.h"
//
// parrot_optimizer_animated.cu
//
// Code generation for function 'parrot_optimizer_animated'
//

// Include files
#include "parrot_optimizer_animated.h"
#include "eml_mtimes_helper.h"
#include "parrot_optimizer_animated_data.h"
#include "parrot_optimizer_animated_emxutil.h"
#include "parrot_optimizer_animated_types.h"
#include "rt_nonfinite.h"
#include "MWCudaDimUtility.hpp"
#include "MWCudaMemoryFunctions.hpp"
#include "MWLaunchParametersUtilities.hpp"
#include "stdio.h"
#include "stdlib.h"
#include "string.h"
#include <cmath>
#include <cstring>

// Variable Definitions
static emlrtRTEInfo emlrtRTEI{
    103,                                                            // lineNo
    24,                                                             // colNo
    "rand",                                                         // fName
    "D:\\Matlab\\fulll\\toolbox\\eml\\lib\\matlab\\randfun\\rand.m" // pName
};

static emlrtRTEInfo b_emlrtRTEI{
    3,                           // lineNo
    5,                           // colNo
    "parrot_optimizer_animated", // fName
    "D:\\GIT_resp\\Optimization-Algorithms\\po\\function\\parrot_optimizer_"
    "animated.m" // pName
};

static emlrtRTEInfo c_emlrtRTEI{
    4,                           // lineNo
    15,                          // colNo
    "parrot_optimizer_animated", // fName
    "D:\\GIT_resp\\Optimization-Algorithms\\po\\function\\parrot_optimizer_"
    "animated.m" // pName
};

static emlrtRTEInfo d_emlrtRTEI{
    6,                           // lineNo
    5,                           // colNo
    "parrot_optimizer_animated", // fName
    "D:\\GIT_resp\\Optimization-Algorithms\\po\\function\\parrot_optimizer_"
    "animated.m" // pName
};

static emlrtRTEInfo e_emlrtRTEI{
    10,                          // lineNo
    25,                          // colNo
    "parrot_optimizer_animated", // fName
    "D:\\GIT_resp\\Optimization-Algorithms\\po\\function\\parrot_optimizer_"
    "animated.m" // pName
};

static emlrtRTEInfo f_emlrtRTEI{
    7,                           // lineNo
    35,                          // colNo
    "parrot_optimizer_animated", // fName
    "D:\\GIT_resp\\Optimization-Algorithms\\po\\function\\parrot_optimizer_"
    "animated.m" // pName
};

static emlrtRTEInfo g_emlrtRTEI{
    181,                     // lineNo
    24,                      // colNo
    "combineVectorElements", // fName
    "D:"
    "\\Matlab\\fulll\\toolbox\\eml\\lib\\matlab\\datafun\\private\\combineVecto"
    "rElements.m" // pName
};

static emlrtRTEInfo h_emlrtRTEI{
    108,                     // lineNo
    13,                      // colNo
    "combineVectorElements", // fName
    "D:"
    "\\Matlab\\fulll\\toolbox\\eml\\lib\\matlab\\datafun\\private\\combineVecto"
    "rElements.m" // pName
};

static emlrtRTEInfo i_emlrtRTEI{
    14,                          // lineNo
    9,                           // colNo
    "parrot_optimizer_animated", // fName
    "D:\\GIT_resp\\Optimization-Algorithms\\po\\function\\parrot_optimizer_"
    "animated.m" // pName
};

static emlrtRTEInfo j_emlrtRTEI{
    15,                          // lineNo
    9,                           // colNo
    "parrot_optimizer_animated", // fName
    "D:\\GIT_resp\\Optimization-Algorithms\\po\\function\\parrot_optimizer_"
    "animated.m" // pName
};

static emlrtRTEInfo k_emlrtRTEI{
    139,                                                             // lineNo
    20,                                                              // colNo
    "randn",                                                         // fName
    "D:\\Matlab\\fulll\\toolbox\\eml\\lib\\matlab\\randfun\\randn.m" // pName
};

static emlrtRTEInfo l_emlrtRTEI{
    75,                          // lineNo
    13,                          // colNo
    "parrot_optimizer_animated", // fName
    "D:\\GIT_resp\\Optimization-Algorithms\\po\\function\\parrot_optimizer_"
    "animated.m" // pName
};

static emlrtRTEInfo m_emlrtRTEI{
    57,                          // lineNo
    17,                          // colNo
    "parrot_optimizer_animated", // fName
    "D:\\GIT_resp\\Optimization-Algorithms\\po\\function\\parrot_optimizer_"
    "animated.m" // pName
};

static emlrtRTEInfo o_emlrtRTEI{
    30,                    // lineNo
    21,                    // colNo
    "applyScalarFunction", // fName
    "D:\\Matlab\\fulll\\toolbox\\eml\\eml\\+coder\\+"
    "internal\\applyScalarFunction.m" // pName
};

static emlrtRTEInfo p_emlrtRTEI{
    129,                         // lineNo
    6,                           // colNo
    "applyBinaryScalarFunction", // fName
    "D:\\Matlab\\fulll\\toolbox\\eml\\eml\\+coder\\+"
    "internal\\applyBinaryScalarFunction.m" // pName
};

static emlrtRTEInfo
    q_emlrtRTEI{
        34,               // lineNo
        1,                // colNo
        "rdivide_helper", // fName
        "D:\\Matlab\\fulll\\toolbox\\eml\\eml\\+coder\\+internal\\rdivide_"
        "helper.m" // pName
    };

static emlrtRTEInfo r_emlrtRTEI{
    40,                          // lineNo
    17,                          // colNo
    "parrot_optimizer_animated", // fName
    "D:\\GIT_resp\\Optimization-Algorithms\\po\\function\\parrot_optimizer_"
    "animated.m" // pName
};

static emlrtRTEInfo s_emlrtRTEI{
    30,                          // lineNo
    17,                          // colNo
    "parrot_optimizer_animated", // fName
    "D:\\GIT_resp\\Optimization-Algorithms\\po\\function\\parrot_optimizer_"
    "animated.m" // pName
};

static emlrtRTEInfo t_emlrtRTEI{
    49,                          // lineNo
    21,                          // colNo
    "parrot_optimizer_animated", // fName
    "D:\\GIT_resp\\Optimization-Algorithms\\po\\function\\parrot_optimizer_"
    "animated.m" // pName
};

static emlrtRTEInfo u_emlrtRTEI{
    3,                // lineNo
    9,                // colNo
    "initialization", // fName
    "D:\\GIT_resp\\Optimization-Algorithms\\po\\function\\initialization.m" // pName
};

static emlrtRTEInfo v_emlrtRTEI{
    4,                           // lineNo
    5,                           // colNo
    "parrot_optimizer_animated", // fName
    "D:\\GIT_resp\\Optimization-Algorithms\\po\\function\\parrot_optimizer_"
    "animated.m" // pName
};

static emlrtRTEInfo w_emlrtRTEI{
    49,                                                             // lineNo
    9,                                                              // colNo
    "mean",                                                         // fName
    "D:\\Matlab\\fulll\\toolbox\\eml\\lib\\matlab\\datafun\\mean.m" // pName
};

static emlrtRTEInfo x_emlrtRTEI{
    13,        // lineNo
    1,         // colNo
    "normrnd", // fName
    "D:\\Matlab\\fulll\\toolbox\\eml\\eml\\+coder\\+internal\\private\\normrnd."
    "m" // pName
};

static emlrtRTEInfo y_emlrtRTEI{
    48,                          // lineNo
    36,                          // colNo
    "parrot_optimizer_animated", // fName
    "D:\\GIT_resp\\Optimization-Algorithms\\po\\function\\parrot_optimizer_"
    "animated.m" // pName
};

static emlrtRTEInfo ab_emlrtRTEI{
    26,                          // lineNo
    17,                          // colNo
    "parrot_optimizer_animated", // fName
    "D:\\GIT_resp\\Optimization-Algorithms\\po\\function\\parrot_optimizer_"
    "animated.m" // pName
};

static emlrtRTEInfo bb_emlrtRTEI{
    48,                          // lineNo
    60,                          // colNo
    "parrot_optimizer_animated", // fName
    "D:\\GIT_resp\\Optimization-Algorithms\\po\\function\\parrot_optimizer_"
    "animated.m" // pName
};

static emlrtRTEInfo cb_emlrtRTEI{
    38,                          // lineNo
    28,                          // colNo
    "parrot_optimizer_animated", // fName
    "D:\\GIT_resp\\Optimization-Algorithms\\po\\function\\parrot_optimizer_"
    "animated.m" // pName
};

static emlrtRTEInfo db_emlrtRTEI{
    27,                          // lineNo
    28,                          // colNo
    "parrot_optimizer_animated", // fName
    "D:\\GIT_resp\\Optimization-Algorithms\\po\\function\\parrot_optimizer_"
    "animated.m" // pName
};

static emlrtRTEInfo eb_emlrtRTEI{
    48,                          // lineNo
    30,                          // colNo
    "parrot_optimizer_animated", // fName
    "D:\\GIT_resp\\Optimization-Algorithms\\po\\function\\parrot_optimizer_"
    "animated.m" // pName
};

static emlrtRTEInfo fb_emlrtRTEI{
    61,                          // lineNo
    25,                          // colNo
    "parrot_optimizer_animated", // fName
    "D:\\GIT_resp\\Optimization-Algorithms\\po\\function\\parrot_optimizer_"
    "animated.m" // pName
};

static emlrtRTEInfo gb_emlrtRTEI{
    40,                          // lineNo
    25,                          // colNo
    "parrot_optimizer_animated", // fName
    "D:\\GIT_resp\\Optimization-Algorithms\\po\\function\\parrot_optimizer_"
    "animated.m" // pName
};

// Function Declarations
static void binary_expand_op(emxArray_real_T *in1, const emxArray_real_T *in2,
                             int32_T in3, const emxArray_real_T *in4,
                             const emxArray_real_T *in5,
                             const emxArray_real_T *in6, real_T in7,
                             const emxArray_real_T *in8);

static void binary_expand_op_1(emxArray_real_T *in1, const emxArray_real_T *in2,
                               int32_T in3, const emxArray_real_T *in4,
                               const emxArray_real_T *in5,
                               const emxArray_real_T *in6,
                               const emxArray_real_T *in7);

static void binary_expand_op_3(emxArray_real_T *in1, const emxArray_real_T *in2,
                               int32_T in3, const emxArray_real_T *in4,
                               const emxArray_real_T *in5);

static void binary_expand_op_4(emxArray_real_T *in1, const emxArray_real_T *in2,
                               int32_T in3, real_T in4,
                               const emxArray_real_T *in5, real_T in6);

static __global__ void c_parrot_optimizer_animated_ker(const real_T lb,
                                                       const real_T St,
                                                       const emxArray_real_T r,
                                                       const int32_T b_r,
                                                       emxArray_real_T X);

static void checkCudaError(hipError_t errCode, const char_T *file,
                           uint32_T b_line);

static uint64_T computeNumIters(int32_T ub);

static __global__ void d_parrot_optimizer_animated_ker(const emxArray_real_T X,
                                                       const int32_T b,
                                                       emxArray_real_T fitness,
                                                       int32_T X_dim0);

static __global__ void e_parrot_optimizer_animated_ker(const emxArray_real_T X,
                                                       const int32_T npages,
                                                       const int32_T b_X,
                                                       emxArray_real_T X_best,
                                                       int32_T X_dim0);

static __global__ void f_parrot_optimizer_animated_ker(const int32_T sz,
                                                       emxArray_real_T y);

static __global__ void g_parrot_optimizer_animated_ker(const int32_T npages,
                                                       const emxArray_real_T y,
                                                       const int32_T b_y,
                                                       emxArray_real_T X_mean);

static void gpuEmxEnsureCapacity_real_T(const emxArray_real_T *cpu,
                                        emxArray_real_T *gpu,
                                        boolean_T needsCopy);

static void gpuEmxFree_real_T(emxArray_real_T *gpu);

static void gpuEmxMemcpyCpuToGpu_real_T(emxArray_real_T *gpu,
                                        const emxArray_real_T *cpu);

static void gpuEmxMemcpyGpuToCpu_real_T(emxArray_real_T *cpu,
                                        emxArray_real_T *gpu);

static void gpuEmxReset_real_T(emxArray_real_T *gpu);

static __global__ void h_parrot_optimizer_animated_ker(const emxArray_real_T v,
                                                       const int32_T b,
                                                       emxArray_real_T y);

static __global__ void i_parrot_optimizer_animated_ker(const emxArray_real_T y,
                                                       const int32_T b,
                                                       emxArray_real_T b_y);

static __global__ void j_parrot_optimizer_animated_ker(
    const emxArray_real_T X_mean, const real_T t2, const emxArray_real_T y,
    const emxArray_real_T r, const emxArray_real_T X_best,
    const emxArray_real_T X, const int32_T i, const int32_T b_X,
    emxArray_real_T X_new, int32_T X_dim0);

static __global__ void
k_parrot_optimizer_animated_ker(const real_T lb, const emxArray_real_T X_new,
                                const int32_T b, emxArray_real_T maxval);

static __global__ void
l_parrot_optimizer_animated_ker(const real_T ub, const emxArray_real_T maxval,
                                const int32_T b, emxArray_real_T X_new);

static __global__ void
m_parrot_optimizer_animated_ker(const emxArray_real_T X_new, const int32_T i,
                                const int32_T b, emxArray_real_T X,
                                int32_T X_dim0);

static __global__ void n_parrot_optimizer_animated_ker(const emxArray_real_T v,
                                                       const int32_T b,
                                                       emxArray_real_T y);

static __global__ void o_parrot_optimizer_animated_ker(const emxArray_real_T y,
                                                       const int32_T b,
                                                       emxArray_real_T b_y);

static __global__ void p_parrot_optimizer_animated_ker(
    const emxArray_real_T dv5, const emxArray_real_T y, const emxArray_real_T r,
    const emxArray_real_T X_best, const emxArray_real_T X, const int32_T i,
    const int32_T b_X, emxArray_real_T X_new, int32_T X_dim0);

static __global__ void q_parrot_optimizer_animated_ker(
    const emxArray_real_T X_mean, const emxArray_real_T X, const int32_T i,
    const real_T t2, const int32_T b_X, emxArray_real_T X_new, int32_T X_dim0);

static __global__ void r_parrot_optimizer_animated_ker(const real_T Max_iter,
                                                       const emxArray_real_T r,
                                                       const int32_T npages,
                                                       const int32_T b_r,
                                                       emxArray_real_T dv4);

static void raiseCudaError(int32_T errCode, const char_T *file, uint32_T b_line,
                           const char_T *errorName, const char_T *errorString);

static real_T rt_powd_snf(real_T u0, real_T u1);

static __global__ void s_parrot_optimizer_animated_ker(const int32_T b,
                                                       emxArray_real_T dv4);

static __global__ void t_parrot_optimizer_animated_ker(
    const emxArray_real_T dv4, const emxArray_real_T r, const emxArray_real_T X,
    const int32_T i, const int32_T b_X, emxArray_real_T X_new, int32_T X_dim0);

static __global__ void
u_parrot_optimizer_animated_ker(const real_T St, const real_T t2,
                                const emxArray_real_T X, const int32_T i,
                                const emxArray_real_T X_best, const int32_T b_X,
                                emxArray_real_T X_new, int32_T X_dim0);

static __global__ void v_parrot_optimizer_animated_ker(const emxArray_real_T X,
                                                       const int32_T npages,
                                                       const int32_T b_X,
                                                       emxArray_real_T X_best,
                                                       int32_T X_dim0);

static __global__ void w_parrot_optimizer_animated_ker(const int32_T vlen,
                                                       const emxArray_real_T X,
                                                       const int32_T b,
                                                       emxArray_real_T y,
                                                       int32_T X_dim0);

// Function Definitions
static void binary_expand_op(emxArray_real_T *in1, const emxArray_real_T *in2,
                             int32_T in3, const emxArray_real_T *in4,
                             const emxArray_real_T *in5,
                             const emxArray_real_T *in6, real_T in7,
                             const emxArray_real_T *in8)
{
  int32_T b;
  int32_T i;
  int32_T stride_0_1;
  int32_T stride_1_1;
  int32_T stride_2_1;
  int32_T stride_3_1;
  int32_T stride_4_1;
  i = in1->size[0] * in1->size[1];
  in1->size[0] = 1;
  if (in8->size[1] == 1) {
    if (in6->size[1] == 1) {
      b = in5->size[1];
    } else {
      b = in6->size[1];
    }
    if (b == 1) {
      if (in4->size[1] == 1) {
        in1->size[1] = in2->size[1];
      } else {
        in1->size[1] = in4->size[1];
      }
    } else if (in6->size[1] == 1) {
      in1->size[1] = in5->size[1];
    } else {
      in1->size[1] = in6->size[1];
    }
  } else {
    in1->size[1] = in8->size[1];
  }
  emxEnsureCapacity_real_T(in1, i, &s_emlrtRTEI);
  stride_0_1 = (in2->size[1] != 1);
  stride_1_1 = (in4->size[1] != 1);
  stride_2_1 = (in5->size[1] != 1);
  stride_3_1 = (in6->size[1] != 1);
  stride_4_1 = (in8->size[1] != 1);
  if (in6->size[1] == 1) {
    b = in5->size[1];
  } else {
    b = in6->size[1];
  }
  if (in8->size[1] == 1) {
    if (b == 1) {
      if (in4->size[1] == 1) {
        b = in2->size[1];
      } else {
        b = in4->size[1];
      }
    } else if (in6->size[1] == 1) {
      b = in5->size[1];
    } else {
      b = in6->size[1];
    }
  } else {
    b = in8->size[1];
  }
  for (i = 0; i < b; i++) {
    in1->data[i] = (in2->data[in3 + in2->size[0] * (i * stride_0_1)] -
                    in4->data[i * stride_1_1]) *
                       (in5->data[i * stride_2_1] * 0.69657450255769682 /
                        in6->data[i * stride_3_1]) +
                   in7 * in8->data[i * stride_4_1];
  }
}

static void binary_expand_op_1(emxArray_real_T *in1, const emxArray_real_T *in2,
                               int32_T in3, const emxArray_real_T *in4,
                               const emxArray_real_T *in5,
                               const emxArray_real_T *in6,
                               const emxArray_real_T *in7)
{
  int32_T b;
  int32_T i;
  int32_T stride_0_1;
  int32_T stride_1_1;
  int32_T stride_2_1;
  int32_T stride_3_1;
  int32_T stride_4_1;
  i = in1->size[0] * in1->size[1];
  in1->size[0] = 1;
  if (in7->size[1] == 1) {
    if (in6->size[1] == 1) {
      b = in5->size[1];
    } else {
      b = in6->size[1];
    }
    if (b == 1) {
      b = in4->size[1];
    } else if (in6->size[1] == 1) {
      b = in5->size[1];
    } else {
      b = in6->size[1];
    }
    if (b == 1) {
      in1->size[1] = in2->size[1];
    } else {
      if (in6->size[1] == 1) {
        b = in5->size[1];
      } else {
        b = in6->size[1];
      }
      if (b == 1) {
        in1->size[1] = in4->size[1];
      } else if (in6->size[1] == 1) {
        in1->size[1] = in5->size[1];
      } else {
        in1->size[1] = in6->size[1];
      }
    }
  } else {
    in1->size[1] = in7->size[1];
  }
  emxEnsureCapacity_real_T(in1, i, &r_emlrtRTEI);
  stride_0_1 = (in2->size[1] != 1);
  stride_1_1 = (in4->size[1] != 1);
  stride_2_1 = (in5->size[1] != 1);
  stride_3_1 = (in6->size[1] != 1);
  stride_4_1 = (in7->size[1] != 1);
  if (in6->size[1] == 1) {
    b = in5->size[1];
  } else {
    b = in6->size[1];
  }
  if (b == 1) {
    b = in4->size[1];
  } else if (in6->size[1] == 1) {
    b = in5->size[1];
  } else {
    b = in6->size[1];
  }
  if (in6->size[1] == 1) {
    i = in5->size[1];
  } else {
    i = in6->size[1];
  }
  if (in7->size[1] == 1) {
    if (b == 1) {
      b = in2->size[1];
    } else if (i == 1) {
      b = in4->size[1];
    } else if (in6->size[1] == 1) {
      b = in5->size[1];
    } else {
      b = in6->size[1];
    }
  } else {
    b = in7->size[1];
  }
  for (i = 0; i < b; i++) {
    in1->data[i] = (in2->data[in3 + in2->size[0] * (i * stride_0_1)] +
                    in4->data[i * stride_1_1] *
                        (in5->data[i * stride_2_1] * 0.69657450255769682 /
                         in6->data[i * stride_3_1])) +
                   in7->data[i * stride_4_1];
  }
}

static void binary_expand_op_3(emxArray_real_T *in1, const emxArray_real_T *in2,
                               int32_T in3, const emxArray_real_T *in4,
                               const emxArray_real_T *in5)
{
  int32_T b;
  int32_T i;
  int32_T stride_0_1;
  int32_T stride_1_1;
  int32_T stride_2_1;
  i = in1->size[0] * in1->size[1];
  in1->size[0] = 1;
  if (in5->size[1] == 1) {
    b = in4->size[1];
  } else {
    b = in5->size[1];
  }
  if (b == 1) {
    in1->size[1] = in2->size[1];
  } else if (in5->size[1] == 1) {
    in1->size[1] = in4->size[1];
  } else {
    in1->size[1] = in5->size[1];
  }
  emxEnsureCapacity_real_T(in1, i, &t_emlrtRTEI);
  stride_0_1 = (in2->size[1] != 1);
  stride_1_1 = (in4->size[1] != 1);
  stride_2_1 = (in5->size[1] != 1);
  if (in5->size[1] == 1) {
    b = in4->size[1];
  } else {
    b = in5->size[1];
  }
  if (b == 1) {
    b = in2->size[1];
  } else if (in5->size[1] == 1) {
    b = in4->size[1];
  } else {
    b = in5->size[1];
  }
  for (i = 0; i < b; i++) {
    in1->data[i] = in2->data[in3 + in2->size[0] * (i * stride_0_1)] +
                   0.2 * in4->data[i * stride_1_1] * in5->data[i * stride_2_1];
  }
}

static void binary_expand_op_4(emxArray_real_T *in1, const emxArray_real_T *in2,
                               int32_T in3, real_T in4,
                               const emxArray_real_T *in5, real_T in6)
{
  int32_T b;
  int32_T i;
  int32_T stride_0_1;
  int32_T stride_1_1;
  int32_T stride_2_1;
  int32_T stride_3_1;
  int32_T stride_4_1;
  i = in1->size[0] * in1->size[1];
  in1->size[0] = 1;
  if (in5->size[1] == 1) {
    b = in2->size[1];
  } else {
    b = in5->size[1];
  }
  if (b == 1) {
    if (in2->size[1] == 1) {
      b = in5->size[1];
    } else {
      b = in2->size[1];
    }
    if (b == 1) {
      in1->size[1] = in2->size[1];
    } else if (in2->size[1] == 1) {
      in1->size[1] = in5->size[1];
    } else {
      in1->size[1] = in2->size[1];
    }
  } else if (in5->size[1] == 1) {
    in1->size[1] = in2->size[1];
  } else {
    in1->size[1] = in5->size[1];
  }
  emxEnsureCapacity_real_T(in1, i, &m_emlrtRTEI);
  stride_0_1 = (in2->size[1] != 1);
  stride_1_1 = (in5->size[1] != 1);
  stride_2_1 = (in2->size[1] != 1);
  stride_3_1 = (in2->size[1] != 1);
  stride_4_1 = (in5->size[1] != 1);
  if (in5->size[1] == 1) {
    b = in2->size[1];
  } else {
    b = in5->size[1];
  }
  if (in2->size[1] == 1) {
    i = in5->size[1];
  } else {
    i = in2->size[1];
  }
  if (b == 1) {
    if (i == 1) {
      b = in2->size[1];
    } else if (in2->size[1] == 1) {
      b = in5->size[1];
    } else {
      b = in2->size[1];
    }
  } else if (in5->size[1] == 1) {
    b = in2->size[1];
  } else {
    b = in5->size[1];
  }
  for (i = 0; i < b; i++) {
    in1->data[i] = (in2->data[in3 + in2->size[0] * (i * stride_0_1)] +
                    in4 * (in5->data[i * stride_1_1] -
                           in2->data[in3 + in2->size[0] * (i * stride_2_1)])) -
                   in6 * (in2->data[in3 + in2->size[0] * (i * stride_3_1)] -
                          in5->data[i * stride_4_1]);
  }
}

static __global__
    __launch_bounds__(1024, 1) void c_parrot_optimizer_animated_ker(
        const real_T lb, const real_T St, const emxArray_real_T r,
        const int32_T b_r, emxArray_real_T X)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_r);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T xpageoffset;
    xpageoffset = static_cast<int32_T>(idx);
    X.data[xpageoffset] = r.data[xpageoffset] * St + lb;
  }
}

static void checkCudaError(hipError_t errCode, const char_T *file,
                           uint32_T b_line)
{
  if (errCode != hipSuccess) {
    raiseCudaError(errCode, file, b_line, hipGetErrorName(errCode),
                   hipGetErrorString(errCode));
  }
}

static uint64_T computeNumIters(int32_T ub)
{
  uint64_T numIters;
  numIters = 0ULL;
  if (ub >= 0) {
    numIters = static_cast<uint64_T>(ub + 1);
  }
  return numIters;
}

static __global__ __launch_bounds__(
    1024, 1) void d_parrot_optimizer_animated_ker(const emxArray_real_T X,
                                                  const int32_T b,
                                                  emxArray_real_T fitness,
                                                  int32_T X_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    real_T d;
    real_T d1;
    real_T t1;
    real_T t2;
    int32_T k;
    k = static_cast<int32_T>(idx);
    t1 = 0.0;
    t2 = 0.0;
    d = X.data[k];
    d1 = X.data[k + X_dim0];
    for (int32_T i{0}; i < 5; i++) {
      t1 += (static_cast<real_T>(i) + 1.0) *
            cos(((static_cast<real_T>(i) + 1.0) + 1.0) * d +
                (static_cast<real_T>(i) + 1.0));
      t2 += (static_cast<real_T>(i) + 1.0) *
            cos(((static_cast<real_T>(i) + 1.0) + 1.0) * d1 +
                (static_cast<real_T>(i) + 1.0));
    }
    fitness.data[k] = t1 * t2;
  }
}

static __global__
    __launch_bounds__(1024, 1) void e_parrot_optimizer_animated_ker(
        const emxArray_real_T X, const int32_T npages, const int32_T b_X,
        emxArray_real_T X_best, int32_T X_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_X);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T xpageoffset;
    xpageoffset = static_cast<int32_T>(idx);
    X_best.data[xpageoffset] = X.data[npages + X_dim0 * xpageoffset];
  }
}

static __global__
    __launch_bounds__(1024,
                      1) void f_parrot_optimizer_animated_ker(const int32_T sz,
                                                              emxArray_real_T y)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(sz);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T xpageoffset;
    xpageoffset = static_cast<int32_T>(idx);
    y.data[xpageoffset] = 0.0;
  }
}

static __global__ __launch_bounds__(
    1024, 1) void g_parrot_optimizer_animated_ker(const int32_T npages,
                                                  const emxArray_real_T y,
                                                  const int32_T b_y,
                                                  emxArray_real_T X_mean)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_y);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T xpageoffset;
    xpageoffset = static_cast<int32_T>(idx);
    X_mean.data[xpageoffset] =
        y.data[xpageoffset] / static_cast<real_T>(npages);
  }
}

static void gpuEmxEnsureCapacity_real_T(const emxArray_real_T *cpu,
                                        emxArray_real_T *gpu,
                                        boolean_T needsCopy)
{
  real_T *newData;
  int32_T i;
  int32_T totalSizeCpu;
  int32_T totalSizeGpu;
  if (gpu->numDimensions == 0) {
    gpu->numDimensions = cpu->numDimensions;
    gpu->size = static_cast<int32_T *>(emlrtCallocMex(
        static_cast<uint32_T>(gpu->numDimensions), sizeof(int32_T)));
  }
  totalSizeCpu = 1;
  totalSizeGpu = 1;
  for (i = 0; i < cpu->numDimensions; i++) {
    totalSizeGpu *= gpu->size[i];
    totalSizeCpu *= cpu->size[i];
    gpu->size[i] = cpu->size[i];
  }
  if (gpu->allocatedSize < totalSizeCpu) {
    i = cpu->allocatedSize;
    if (i < totalSizeCpu) {
      i = totalSizeCpu;
    }
    checkCudaError(
        mwCudaMalloc(&newData, static_cast<uint32_T>(i) * sizeof(real_T)),
        __FILE__, __LINE__);
    needsCopy = (needsCopy && gpu->canFreeData);
    if (needsCopy) {
      checkCudaError(
          hipMemcpy(newData, gpu->data,
                     static_cast<uint32_T>(totalSizeGpu) * sizeof(real_T),
                     hipMemcpyDeviceToDevice),
          __FILE__, __LINE__);
    }
    if (gpu->canFreeData) {
      checkCudaError(mwCudaFree(gpu->data), __FILE__, __LINE__);
    }
    gpu->data = newData;
    gpu->allocatedSize = i;
    gpu->canFreeData = true;
  }
}

static void gpuEmxFree_real_T(emxArray_real_T *gpu)
{
  if (gpu->data != (void *)4207599121ULL) {
    checkCudaError(mwCudaFree(gpu->data), __FILE__, __LINE__);
  }
  emlrtFreeMex(gpu->size);
}

static void gpuEmxMemcpyCpuToGpu_real_T(emxArray_real_T *gpu,
                                        const emxArray_real_T *cpu)
{
  int32_T actualSize;
  actualSize = 1;
  for (int32_T i{0}; i < cpu->numDimensions; i++) {
    actualSize *= cpu->size[i];
  }
  checkCudaError(hipMemcpy(gpu->data, cpu->data,
                            static_cast<uint32_T>(actualSize) * sizeof(real_T),
                            hipMemcpyHostToDevice),
                 __FILE__, __LINE__);
}

static void gpuEmxMemcpyGpuToCpu_real_T(emxArray_real_T *cpu,
                                        emxArray_real_T *gpu)
{
  int32_T actualSize;
  actualSize = 1;
  for (int32_T i{0}; i < cpu->numDimensions; i++) {
    actualSize *= cpu->size[i];
  }
  checkCudaError(hipMemcpy(cpu->data, gpu->data,
                            static_cast<uint32_T>(actualSize) * sizeof(real_T),
                            hipMemcpyDeviceToHost),
                 __FILE__, __LINE__);
}

static void gpuEmxReset_real_T(emxArray_real_T *gpu)
{
  std::memset(gpu, 0, sizeof(emxArray_real_T));
}

static __global__
    __launch_bounds__(1024, 1) void h_parrot_optimizer_animated_ker(
        const emxArray_real_T v, const int32_T b, emxArray_real_T y)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T k;
    k = static_cast<int32_T>(idx);
    y.data[k] = fabs(v.data[k]);
  }
}

static __global__
    __launch_bounds__(1024, 1) void i_parrot_optimizer_animated_ker(
        const emxArray_real_T y, const int32_T b, emxArray_real_T b_y)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T k;
    k = static_cast<int32_T>(idx);
    b_y.data[k] = pow(y.data[k], 0.66666666666666663);
  }
}

static __global__
    __launch_bounds__(1024, 1) void j_parrot_optimizer_animated_ker(
        const emxArray_real_T X_mean, const real_T t2, const emxArray_real_T y,
        const emxArray_real_T r, const emxArray_real_T X_best,
        const emxArray_real_T X, const int32_T i, const int32_T b_X,
        emxArray_real_T X_new, int32_T X_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_X);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T xpageoffset;
    xpageoffset = static_cast<int32_T>(idx);
    X_new.data[xpageoffset] =
        (X.data[i + X_dim0 * xpageoffset] - X_best.data[xpageoffset]) *
            (r.data[xpageoffset] * 0.69657450255769682 / y.data[xpageoffset]) +
        t2 * X_mean.data[xpageoffset];
  }
}

static __global__ __launch_bounds__(
    1024, 1) void k_parrot_optimizer_animated_ker(const real_T lb,
                                                  const emxArray_real_T X_new,
                                                  const int32_T b,
                                                  emxArray_real_T maxval)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T k;
    k = static_cast<int32_T>(idx);
    maxval.data[k] = fmax(X_new.data[k], lb);
  }
}

static __global__ __launch_bounds__(
    1024, 1) void l_parrot_optimizer_animated_ker(const real_T ub,
                                                  const emxArray_real_T maxval,
                                                  const int32_T b,
                                                  emxArray_real_T X_new)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T k;
    k = static_cast<int32_T>(idx);
    X_new.data[k] = fmin(maxval.data[k], ub);
  }
}

static __global__
    __launch_bounds__(1024, 1) void m_parrot_optimizer_animated_ker(
        const emxArray_real_T X_new, const int32_T i, const int32_T b,
        emxArray_real_T X, int32_T X_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T xpageoffset;
    xpageoffset = static_cast<int32_T>(idx);
    X.data[i + X_dim0 * xpageoffset] = X_new.data[xpageoffset];
  }
}

static __global__
    __launch_bounds__(1024, 1) void n_parrot_optimizer_animated_ker(
        const emxArray_real_T v, const int32_T b, emxArray_real_T y)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T k;
    k = static_cast<int32_T>(idx);
    y.data[k] = fabs(v.data[k]);
  }
}

static __global__
    __launch_bounds__(1024, 1) void o_parrot_optimizer_animated_ker(
        const emxArray_real_T y, const int32_T b, emxArray_real_T b_y)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T k;
    k = static_cast<int32_T>(idx);
    b_y.data[k] = pow(y.data[k], 0.66666666666666663);
  }
}

static __global__
    __launch_bounds__(1024, 1) void p_parrot_optimizer_animated_ker(
        const emxArray_real_T dv5, const emxArray_real_T y,
        const emxArray_real_T r, const emxArray_real_T X_best,
        const emxArray_real_T X, const int32_T i, const int32_T b_X,
        emxArray_real_T X_new, int32_T X_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_X);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T xpageoffset;
    xpageoffset = static_cast<int32_T>(idx);
    X_new.data[xpageoffset] =
        (X.data[i + X_dim0 * xpageoffset] +
         X_best.data[xpageoffset] * (r.data[xpageoffset] * 0.69657450255769682 /
                                     y.data[xpageoffset])) +
        dv5.data[xpageoffset];
  }
}

static __global__
    __launch_bounds__(1024, 1) void q_parrot_optimizer_animated_ker(
        const emxArray_real_T X_mean, const emxArray_real_T X, const int32_T i,
        const real_T t2, const int32_T b_X, emxArray_real_T X_new,
        int32_T X_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_X);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T xpageoffset;
    xpageoffset = static_cast<int32_T>(idx);
    X_new.data[xpageoffset] =
        t2 * (X.data[i + X_dim0 * xpageoffset] - X_mean.data[xpageoffset]);
  }
}

static __global__
    __launch_bounds__(1024, 1) void r_parrot_optimizer_animated_ker(
        const real_T Max_iter, const emxArray_real_T r, const int32_T npages,
        const int32_T b_r, emxArray_real_T dv4)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_r);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T xpageoffset;
    xpageoffset = static_cast<int32_T>(idx);
    dv4.data[xpageoffset] =
        static_cast<real_T>(npages) / (r.data[xpageoffset] * Max_iter);
  }
}

static void raiseCudaError(int32_T errCode, const char_T *file, uint32_T b_line,
                           const char_T *errorName, const char_T *errorString)
{
  emlrtRTEInfo rtInfo;
  uint64_T len;
  char_T *brk;
  char_T *fn;
  char_T *pn;
  len = strlen(file);
  pn = static_cast<char_T *>(calloc(len + 1ULL, 1ULL));
  fn = static_cast<char_T *>(calloc(len + 1ULL, 1ULL));
  memcpy(pn, file, len);
  memcpy(fn, file, len);
  brk = strrchr(fn, '.');
  *brk = '\x00';
  brk = strrchr(fn, '/');
  if (brk == nullptr) {
    brk = strrchr(fn, '\\');
  }
  if (brk == nullptr) {
    brk = fn;
  } else {
    brk++;
  }
  rtInfo.lineNo = static_cast<int32_T>(b_line);
  rtInfo.colNo = 0;
  rtInfo.fName = brk;
  rtInfo.pName = pn;
  emlrtCUDAError(static_cast<uint32_T>(errCode), (char_T *)errorName,
                 (char_T *)errorString, &rtInfo, emlrtRootTLSGlobal);
}

static real_T rt_powd_snf(real_T u0, real_T u1)
{
  real_T y;
  if (std::isnan(u0) || std::isnan(u1)) {
    y = rtNaN;
  } else {
    real_T b;
    real_T c;
    b = std::abs(u0);
    c = std::abs(u1);
    if (std::isinf(u1)) {
      if (b == 1.0) {
        y = 1.0;
      } else if (b > 1.0) {
        if (u1 > 0.0) {
          y = rtInf;
        } else {
          y = 0.0;
        }
      } else if (u1 > 0.0) {
        y = 0.0;
      } else {
        y = rtInf;
      }
    } else if (c == 0.0) {
      y = 1.0;
    } else if (c == 1.0) {
      if (u1 > 0.0) {
        y = u0;
      } else {
        y = 1.0 / u0;
      }
    } else if (u1 == 2.0) {
      y = u0 * u0;
    } else if ((u1 == 0.5) && (u0 >= 0.0)) {
      y = std::sqrt(u0);
    } else if ((u0 < 0.0) && (u1 > std::floor(u1))) {
      y = rtNaN;
    } else {
      y = std::pow(u0, u1);
    }
  }
  return y;
}

static __global__ __launch_bounds__(
    1024, 1) void s_parrot_optimizer_animated_ker(const int32_T b,
                                                  emxArray_real_T dv4)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T k;
    k = static_cast<int32_T>(idx);
    dv4.data[k] = exp(dv4.data[k]);
  }
}

static __global__
    __launch_bounds__(1024, 1) void t_parrot_optimizer_animated_ker(
        const emxArray_real_T dv4, const emxArray_real_T r,
        const emxArray_real_T X, const int32_T i, const int32_T b_X,
        emxArray_real_T X_new, int32_T X_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_X);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T xpageoffset;
    xpageoffset = static_cast<int32_T>(idx);
    X_new.data[xpageoffset] = X.data[i + X_dim0 * xpageoffset] +
                              0.2 * r.data[xpageoffset] * dv4.data[xpageoffset];
  }
}

static __global__
    __launch_bounds__(1024, 1) void u_parrot_optimizer_animated_ker(
        const real_T St, const real_T t2, const emxArray_real_T X,
        const int32_T i, const emxArray_real_T X_best, const int32_T b_X,
        emxArray_real_T X_new, int32_T X_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_X);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    real_T d;
    int32_T xpageoffset;
    xpageoffset = static_cast<int32_T>(idx);
    d = X_best.data[xpageoffset];
    X_new.data[xpageoffset] = (X.data[i + X_dim0 * xpageoffset] +
                               t2 * (d - X.data[i + X_dim0 * xpageoffset])) -
                              St * (X.data[i + X_dim0 * xpageoffset] - d);
  }
}

static __global__
    __launch_bounds__(1024, 1) void v_parrot_optimizer_animated_ker(
        const emxArray_real_T X, const int32_T npages, const int32_T b_X,
        emxArray_real_T X_best, int32_T X_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_X);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T xpageoffset;
    xpageoffset = static_cast<int32_T>(idx);
    X_best.data[xpageoffset] = X.data[npages + X_dim0 * xpageoffset];
  }
}

static __global__
    __launch_bounds__(1024, 1) void w_parrot_optimizer_animated_ker(
        const int32_T vlen, const emxArray_real_T X, const int32_T b,
        emxArray_real_T y, int32_T X_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i;
    int32_T xpageoffset;
    i = static_cast<int32_T>(idx);
    xpageoffset = i * X_dim0;
    y.data[i] = X.data[xpageoffset];
    for (int32_T k{0}; k <= vlen - 2; k++) {
      y.data[i] += X.data[(xpageoffset + k) + 1];
    }
  }
}

void parrot_optimizer_animated(real_T N, real_T Max_iter, real_T lb, real_T ub,
                               real_T dim, emxArray_real_T *cpu_X_best,
                               real_T *f_best,
                               emxArray_real_T *Convergence_curve,
                               emxArray_cell_wrap_0 *all_positions)
{
  dim3 block;
  dim3 grid;
  emxArray_real_T b_gpu_r;
  emxArray_real_T b_gpu_y;
  emxArray_real_T c_gpu_r;
  emxArray_real_T c_gpu_y;
  emxArray_real_T d_gpu_r;
  emxArray_real_T d_gpu_y;
  emxArray_real_T e_gpu_r;
  emxArray_real_T e_gpu_y;
  emxArray_real_T gpu_X;
  emxArray_real_T gpu_X_best;
  emxArray_real_T gpu_X_mean;
  emxArray_real_T gpu_X_new;
  emxArray_real_T gpu_dv4;
  emxArray_real_T gpu_dv5;
  emxArray_real_T gpu_fitness;
  emxArray_real_T gpu_maxval;
  emxArray_real_T gpu_r;
  emxArray_real_T gpu_v;
  emxArray_real_T gpu_y;
  emxArray_real_T *b_cpu_r;
  emxArray_real_T *b_cpu_y;
  emxArray_real_T *c_cpu_r;
  emxArray_real_T *c_cpu_y;
  emxArray_real_T *cpu_X;
  emxArray_real_T *cpu_X_mean;
  emxArray_real_T *cpu_X_new;
  emxArray_real_T *cpu_dv4;
  emxArray_real_T *cpu_dv5;
  emxArray_real_T *cpu_fitness;
  emxArray_real_T *cpu_maxval;
  emxArray_real_T *cpu_r;
  emxArray_real_T *cpu_v;
  emxArray_real_T *cpu_y;
  emxArray_real_T *d_cpu_r;
  emxArray_real_T *d_cpu_y;
  emxArray_real_T *e_cpu_r;
  emxArray_real_T *e_cpu_y;
  real_T St;
  real_T ex;
  int32_T b_i;
  int32_T i;
  int32_T i2;
  int32_T i3;
  int32_T npages;
  int32_T vlen;
  boolean_T X_best_needsGpuEnsureCapacity;
  boolean_T X_best_outdatedOnCpu;
  boolean_T X_needsGpuEnsureCapacity;
  boolean_T X_outdatedOnCpu;
  boolean_T b_r_outdatedOnGpu;
  boolean_T c_r_outdatedOnGpu;
  boolean_T dv5_outdatedOnGpu;
  boolean_T p;
  boolean_T r_outdatedOnGpu;
  boolean_T v_outdatedOnGpu;
  boolean_T validLaunchParams;
  gpuEmxReset_real_T(&gpu_dv5);
  gpuEmxReset_real_T(&c_gpu_y);
  gpuEmxReset_real_T(&e_gpu_y);
  gpuEmxReset_real_T(&gpu_maxval);
  gpuEmxReset_real_T(&gpu_dv4);
  gpuEmxReset_real_T(&b_gpu_y);
  gpuEmxReset_real_T(&d_gpu_y);
  gpuEmxReset_real_T(&d_gpu_r);
  gpuEmxReset_real_T(&gpu_X_new);
  gpuEmxReset_real_T(&gpu_v);
  gpuEmxReset_real_T(&e_gpu_r);
  gpuEmxReset_real_T(&b_gpu_r);
  gpuEmxReset_real_T(&c_gpu_r);
  gpuEmxReset_real_T(&gpu_X_mean);
  gpuEmxReset_real_T(&gpu_y);
  gpuEmxReset_real_T(&gpu_fitness);
  gpuEmxReset_real_T(&gpu_X);
  gpuEmxReset_real_T(&gpu_r);
  gpuEmxReset_real_T(&gpu_X_best);
  dv5_outdatedOnGpu = false;
  v_outdatedOnGpu = false;
  c_r_outdatedOnGpu = false;
  b_r_outdatedOnGpu = false;
  emlrtHeapReferenceStackEnterFcnR2012b(emlrtRootTLSGlobal);
  //  初始化
  //  生成均匀分布的初始种群
  emxInit_real_T(&cpu_r, 2, &u_emlrtRTEI, true);
  r_outdatedOnGpu = false;
  i = cpu_r->size[0] * cpu_r->size[1];
  cpu_r->size[0] = static_cast<int32_T>(N);
  cpu_r->size[1] = static_cast<int32_T>(dim);
  emxEnsureCapacity_real_T(cpu_r, i, &emlrtRTEI);
  if ((static_cast<int32_T>(N) != 0) && (static_cast<int32_T>(dim) != 0)) {
    emlrtRandu(&cpu_r->data[0],
               static_cast<int32_T>(N) * static_cast<int32_T>(dim));
    r_outdatedOnGpu = true;
  }
  emxInit_real_T(&cpu_X, 2, &b_emlrtRTEI, true);
  i = cpu_X->size[0] * cpu_X->size[1];
  cpu_X->size[0] = cpu_r->size[0];
  cpu_X->size[1] = cpu_r->size[1];
  emxEnsureCapacity_real_T(cpu_X, i, &b_emlrtRTEI);
  vlen = cpu_r->size[0] * cpu_r->size[1] - 1;
  mwGetLaunchParameters1D(computeNumIters(vlen), &grid, &block, 2147483647U);
  gpuEmxEnsureCapacity_real_T(cpu_r, &gpu_r, !r_outdatedOnGpu);
  gpuEmxEnsureCapacity_real_T(cpu_X, &gpu_X, true);
  if (r_outdatedOnGpu) {
    gpuEmxMemcpyCpuToGpu_real_T(&gpu_r, cpu_r);
  }
  validLaunchParams = mwValidateLaunchParameters(grid, block);
  if (validLaunchParams) {
    c_parrot_optimizer_animated_ker<<<grid, block>>>(lb, ub - lb, gpu_r, vlen,
                                                     gpu_X);
  }
  hipDeviceSynchronize();
  checkCudaError(hipGetLastError(), __FILE__, __LINE__);
  X_outdatedOnCpu = true;
  emxFree_real_T(&cpu_r);
  //  输入：x为N×2矩阵，每行代表一个个体（二维）
  //  输出：N×1向量，每个个体的适应度值
  emxInit_real_T(&cpu_fitness, 1, &v_emlrtRTEI, true);
  i = cpu_fitness->size[0];
  cpu_fitness->size[0] = cpu_X->size[0];
  emxEnsureCapacity_real_T(cpu_fitness, i, &c_emlrtRTEI);
  b_i = cpu_X->size[0];
  mwGetLaunchParameters1D(computeNumIters(b_i - 1), &grid, &block, 2147483647U);
  gpuEmxEnsureCapacity_real_T(cpu_fitness, &gpu_fitness, true);
  validLaunchParams = mwValidateLaunchParameters(grid, block);
  if (validLaunchParams) {
    d_parrot_optimizer_animated_ker<<<grid, block>>>(
        gpu_X, b_i - 1, gpu_fitness, cpu_X->size[0U]);
  }
  hipDeviceSynchronize();
  checkCudaError(hipGetLastError(), __FILE__, __LINE__);
  npages = 0;
  vlen = cpu_fitness->size[0];
  gpuEmxMemcpyGpuToCpu_real_T(cpu_fitness, &gpu_fitness);
  ex = cpu_fitness->data[0];
  for (b_i = 0; b_i <= vlen - 2; b_i++) {
    if (std::isnan(cpu_fitness->data[b_i + 1])) {
      p = false;
    } else if (std::isnan(ex)) {
      p = true;
    } else {
      p = (ex > cpu_fitness->data[b_i + 1]);
    }
    if (p) {
      ex = cpu_fitness->data[b_i + 1];
      npages = b_i + 1;
    }
  }
  i = cpu_X_best->size[0] * cpu_X_best->size[1];
  cpu_X_best->size[0] = 1;
  cpu_X_best->size[1] = cpu_X->size[1];
  emxEnsureCapacity_real_T(cpu_X_best, i, &d_emlrtRTEI);
  vlen = cpu_X->size[1] - 1;
  mwGetLaunchParameters1D(computeNumIters(vlen), &grid, &block, 2147483647U);
  X_needsGpuEnsureCapacity = false;
  gpuEmxEnsureCapacity_real_T(cpu_X_best, &gpu_X_best, true);
  X_best_needsGpuEnsureCapacity = false;
  validLaunchParams = mwValidateLaunchParameters(grid, block);
  if (validLaunchParams) {
    e_parrot_optimizer_animated_ker<<<grid, block>>>(
        gpu_X, npages, vlen, gpu_X_best, cpu_X->size[0U]);
  }
  hipDeviceSynchronize();
  checkCudaError(hipGetLastError(), __FILE__, __LINE__);
  X_best_outdatedOnCpu = true;
  //  新增轨迹存储
  //  初始化 Convergence_curve 数组
  i = Convergence_curve->size[0];
  Convergence_curve->size[0] = static_cast<int32_T>(Max_iter);
  emxEnsureCapacity_real_T(Convergence_curve, i, &e_emlrtRTEI);
  //  预先分配足够内存
  //  迭代优化
  i2 = static_cast<int32_T>(Max_iter);
  i = all_positions->size[0];
  all_positions->size[0] = static_cast<int32_T>(Max_iter);
  emxEnsureCapacity_cell_wrap_0(all_positions, i, &f_emlrtRTEI);
  emxInit_real_T(&cpu_y, 2, &w_emlrtRTEI, true);
  if (static_cast<int32_T>(Max_iter) - 1 >= 0) {
    i3 = static_cast<int32_T>(N);
  }
  emxInit_real_T(&cpu_X_mean, 2, &i_emlrtRTEI, true);
  emxInit_real_T(&b_cpu_r, 2, &x_emlrtRTEI, true);
  emxInit_real_T(&c_cpu_r, 2, &x_emlrtRTEI, true);
  emxInit_real_T(&d_cpu_r, 2, &y_emlrtRTEI, true);
  emxInit_real_T(&cpu_v, 2, &ab_emlrtRTEI, true);
  emxInit_real_T(&cpu_X_new, 2, &s_emlrtRTEI, true);
  emxInit_real_T(&e_cpu_r, 2, &bb_emlrtRTEI, true);
  emxInit_real_T(&b_cpu_y, 2, &cb_emlrtRTEI, true);
  emxInit_real_T(&c_cpu_y, 2, &db_emlrtRTEI, true);
  emxInit_real_T(&cpu_dv4, 2, &eb_emlrtRTEI, true);
  emxInit_real_T(&cpu_maxval, 2, &fb_emlrtRTEI, true);
  emxInit_real_T(&d_cpu_y, 2, &cb_emlrtRTEI, true);
  emxInit_real_T(&e_cpu_y, 2, &db_emlrtRTEI, true);
  emxInit_real_T(&cpu_dv5, 2, &gb_emlrtRTEI, true);
  for (int32_T t{0}; t < i2; t++) {
    boolean_T X_mean_needsGpuEnsureCapacity;
    boolean_T X_mean_outdatedOnCpu;
    vlen = cpu_X->size[0];
    if ((cpu_X->size[0] == 0) || (cpu_X->size[1] == 0)) {
      uint32_T sz[2];
      for (vlen = 0; vlen < 2; vlen++) {
        sz[vlen] = static_cast<uint32_T>(cpu_X->size[vlen]);
      }
      i = cpu_y->size[0] * cpu_y->size[1];
      cpu_y->size[0] = 1;
      cpu_y->size[1] = static_cast<int32_T>(sz[1]);
      emxEnsureCapacity_real_T(cpu_y, i, &h_emlrtRTEI);
      mwGetLaunchParameters1D(computeNumIters(static_cast<int32_T>(sz[1]) - 1),
                              &grid, &block, 2147483647U);
      gpuEmxEnsureCapacity_real_T(cpu_y, &gpu_y, true);
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        f_parrot_optimizer_animated_ker<<<grid, block>>>(
            static_cast<int32_T>(sz[1]) - 1, gpu_y);
      }
      hipDeviceSynchronize();
      checkCudaError(hipGetLastError(), __FILE__, __LINE__);
    } else {
      npages = cpu_X->size[1];
      i = cpu_y->size[0] * cpu_y->size[1];
      cpu_y->size[0] = 1;
      cpu_y->size[1] = cpu_X->size[1];
      emxEnsureCapacity_real_T(cpu_y, i, &g_emlrtRTEI);
      mwGetLaunchParameters1D(computeNumIters(npages - 1), &grid, &block,
                              2147483647U);
      if (X_needsGpuEnsureCapacity) {
        gpuEmxEnsureCapacity_real_T(cpu_X, &gpu_X, true);
      }
      X_needsGpuEnsureCapacity = false;
      gpuEmxEnsureCapacity_real_T(cpu_y, &gpu_y, true);
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        w_parrot_optimizer_animated_ker<<<grid, block>>>(
            vlen, gpu_X, npages - 1, gpu_y, cpu_X->size[0U]);
      }
      hipDeviceSynchronize();
      checkCudaError(hipGetLastError(), __FILE__, __LINE__);
    }
    i = cpu_X_mean->size[0] * cpu_X_mean->size[1];
    cpu_X_mean->size[0] = 1;
    cpu_X_mean->size[1] = cpu_y->size[1];
    emxEnsureCapacity_real_T(cpu_X_mean, i, &i_emlrtRTEI);
    npages = cpu_X->size[0];
    vlen = cpu_y->size[1] - 1;
    mwGetLaunchParameters1D(computeNumIters(vlen), &grid, &block, 2147483647U);
    gpuEmxEnsureCapacity_real_T(cpu_X_mean, &gpu_X_mean, true);
    X_mean_needsGpuEnsureCapacity = false;
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      g_parrot_optimizer_animated_ker<<<grid, block>>>(npages, gpu_y, vlen,
                                                       gpu_X_mean);
    }
    hipDeviceSynchronize();
    checkCudaError(hipGetLastError(), __FILE__, __LINE__);
    X_mean_outdatedOnCpu = true;
    i = all_positions->data[t].f1->size[0] * all_positions->data[t].f1->size[1];
    all_positions->data[t].f1->size[0] = cpu_X->size[0];
    all_positions->data[t].f1->size[1] = cpu_X->size[1];
    emxEnsureCapacity_real_T(all_positions->data[t].f1, i, &j_emlrtRTEI);
    for (i = 0; i < cpu_X->size[0] * cpu_X->size[1]; i++) {
      if (X_outdatedOnCpu) {
        gpuEmxMemcpyGpuToCpu_real_T(cpu_X, &gpu_X);
      }
      X_outdatedOnCpu = false;
      all_positions->data[t].f1->data[i] = cpu_X->data[i];
      X_needsGpuEnsureCapacity = true;
    }
    //  记录当前代粒子位置
    for (b_i = 0; b_i < i3; b_i++) {
      real_T d2;
      real_T d3;
      real_T t2;
      boolean_T X_new_needsGpuEnsureCapacity;
      //  随机行为选择（同原算法）
      emlrtRandu(&St, 1);
      St = std::floor(St * 4.0);
      //  行为1：觅食行为
      if (St + 1.0 == 1.0) {
        //  Levy指数
        i = c_cpu_r->size[0] * c_cpu_r->size[1];
        c_cpu_r->size[0] = 1;
        c_cpu_r->size[1] = static_cast<int32_T>(dim);
        emxEnsureCapacity_real_T(c_cpu_r, i, &k_emlrtRTEI);
        if (static_cast<int32_T>(dim) != 0) {
          emlrtRandn(&c_cpu_r->data[0], static_cast<int32_T>(dim));
          c_r_outdatedOnGpu = true;
        }
        i = cpu_v->size[0] * cpu_v->size[1];
        cpu_v->size[0] = 1;
        cpu_v->size[1] = static_cast<int32_T>(dim);
        emxEnsureCapacity_real_T(cpu_v, i, &k_emlrtRTEI);
        if (static_cast<int32_T>(dim) != 0) {
          emlrtRandn(&cpu_v->data[0], static_cast<int32_T>(dim));
          v_outdatedOnGpu = true;
        }
        npages = cpu_v->size[1];
        i = c_cpu_y->size[0] * c_cpu_y->size[1];
        c_cpu_y->size[0] = 1;
        c_cpu_y->size[1] = cpu_v->size[1];
        emxEnsureCapacity_real_T(c_cpu_y, i, &o_emlrtRTEI);
        mwGetLaunchParameters1D(computeNumIters(npages - 1), &grid, &block,
                                2147483647U);
        gpuEmxEnsureCapacity_real_T(cpu_v, &gpu_v, !v_outdatedOnGpu);
        gpuEmxEnsureCapacity_real_T(c_cpu_y, &b_gpu_y, true);
        if (v_outdatedOnGpu) {
          gpuEmxMemcpyCpuToGpu_real_T(&gpu_v, cpu_v);
        }
        v_outdatedOnGpu = false;
        validLaunchParams = mwValidateLaunchParameters(grid, block);
        if (validLaunchParams) {
          h_parrot_optimizer_animated_ker<<<grid, block>>>(gpu_v, npages - 1,
                                                           b_gpu_y);
        }
        hipDeviceSynchronize();
        checkCudaError(hipGetLastError(), __FILE__, __LINE__);
        i = e_cpu_y->size[0] * e_cpu_y->size[1];
        e_cpu_y->size[0] = 1;
        e_cpu_y->size[1] = c_cpu_y->size[1];
        emxEnsureCapacity_real_T(e_cpu_y, i, &p_emlrtRTEI);
        npages = c_cpu_y->size[1];
        mwGetLaunchParameters1D(computeNumIters(npages - 1), &grid, &block,
                                2147483647U);
        gpuEmxEnsureCapacity_real_T(e_cpu_y, &c_gpu_y, true);
        validLaunchParams = mwValidateLaunchParameters(grid, block);
        if (validLaunchParams) {
          i_parrot_optimizer_animated_ker<<<grid, block>>>(b_gpu_y, npages - 1,
                                                           c_gpu_y);
        }
        hipDeviceSynchronize();
        checkCudaError(hipGetLastError(), __FILE__, __LINE__);
        //  Levy飞行向量
        emlrtRandu(&St, 1);
        t2 = St * rt_powd_snf(1.0 - (static_cast<real_T>(t) + 1.0) / Max_iter,
                              2.0 * (static_cast<real_T>(t) + 1.0) / Max_iter);
        if (cpu_X->size[1] == 1) {
          vlen = cpu_X_best->size[1];
        } else {
          vlen = cpu_X->size[1];
        }
        if (c_cpu_r->size[1] == 1) {
          i = e_cpu_y->size[1];
        } else {
          i = c_cpu_r->size[1];
        }
        if (cpu_X->size[1] == 1) {
          npages = cpu_X_best->size[1];
        } else {
          npages = cpu_X->size[1];
        }
        if (npages == 1) {
          if (c_cpu_r->size[1] == 1) {
            npages = e_cpu_y->size[1];
          } else {
            npages = c_cpu_r->size[1];
          }
        } else if (cpu_X->size[1] == 1) {
          npages = cpu_X_best->size[1];
        } else {
          npages = cpu_X->size[1];
        }
        if ((cpu_X->size[1] == cpu_X_best->size[1]) &&
            (c_cpu_r->size[1] == e_cpu_y->size[1]) && (vlen == i) &&
            (npages == cpu_X_mean->size[1])) {
          i = cpu_X_new->size[0] * cpu_X_new->size[1];
          cpu_X_new->size[0] = 1;
          cpu_X_new->size[1] = cpu_X->size[1];
          emxEnsureCapacity_real_T(cpu_X_new, i, &s_emlrtRTEI);
          vlen = cpu_X->size[1] - 1;
          mwGetLaunchParameters1D(computeNumIters(vlen), &grid, &block,
                                  2147483647U);
          if (X_mean_needsGpuEnsureCapacity) {
            gpuEmxEnsureCapacity_real_T(cpu_X_mean, &gpu_X_mean, true);
          }
          X_mean_needsGpuEnsureCapacity = false;
          gpuEmxEnsureCapacity_real_T(c_cpu_r, &b_gpu_r, !c_r_outdatedOnGpu);
          if (X_best_needsGpuEnsureCapacity) {
            gpuEmxEnsureCapacity_real_T(cpu_X_best, &gpu_X_best, true);
          }
          X_best_needsGpuEnsureCapacity = false;
          if (X_needsGpuEnsureCapacity) {
            gpuEmxEnsureCapacity_real_T(cpu_X, &gpu_X, true);
          }
          X_needsGpuEnsureCapacity = false;
          gpuEmxEnsureCapacity_real_T(cpu_X_new, &gpu_X_new, true);
          X_new_needsGpuEnsureCapacity = false;
          if (c_r_outdatedOnGpu) {
            gpuEmxMemcpyCpuToGpu_real_T(&b_gpu_r, c_cpu_r);
          }
          c_r_outdatedOnGpu = false;
          validLaunchParams = mwValidateLaunchParameters(grid, block);
          if (validLaunchParams) {
            j_parrot_optimizer_animated_ker<<<grid, block>>>(
                gpu_X_mean, t2, c_gpu_y, b_gpu_r, gpu_X_best, gpu_X, b_i, vlen,
                gpu_X_new, cpu_X->size[0U]);
          }
          hipDeviceSynchronize();
          checkCudaError(hipGetLastError(), __FILE__, __LINE__);
          p = false;
        } else {
          if (X_outdatedOnCpu) {
            gpuEmxMemcpyGpuToCpu_real_T(cpu_X, &gpu_X);
          }
          X_outdatedOnCpu = false;
          if (X_best_outdatedOnCpu) {
            gpuEmxMemcpyGpuToCpu_real_T(cpu_X_best, &gpu_X_best);
          }
          X_best_outdatedOnCpu = false;
          gpuEmxMemcpyGpuToCpu_real_T(e_cpu_y, &c_gpu_y);
          if (X_mean_outdatedOnCpu) {
            gpuEmxMemcpyGpuToCpu_real_T(cpu_X_mean, &gpu_X_mean);
          }
          X_mean_outdatedOnCpu = false;
          binary_expand_op(cpu_X_new, cpu_X, b_i, cpu_X_best, c_cpu_r, e_cpu_y,
                           t2, cpu_X_mean);
          p = true;
          X_mean_needsGpuEnsureCapacity = true;
          X_best_needsGpuEnsureCapacity = true;
          X_needsGpuEnsureCapacity = true;
          X_new_needsGpuEnsureCapacity = true;
        }
        //  行为2：停留行为
      } else if (St + 1.0 == 2.0) {
        i = b_cpu_r->size[0] * b_cpu_r->size[1];
        b_cpu_r->size[0] = 1;
        b_cpu_r->size[1] = static_cast<int32_T>(dim);
        emxEnsureCapacity_real_T(b_cpu_r, i, &k_emlrtRTEI);
        if (static_cast<int32_T>(dim) != 0) {
          emlrtRandn(&b_cpu_r->data[0], static_cast<int32_T>(dim));
          b_r_outdatedOnGpu = true;
        }
        i = cpu_v->size[0] * cpu_v->size[1];
        cpu_v->size[0] = 1;
        cpu_v->size[1] = static_cast<int32_T>(dim);
        emxEnsureCapacity_real_T(cpu_v, i, &k_emlrtRTEI);
        if (static_cast<int32_T>(dim) != 0) {
          emlrtRandn(&cpu_v->data[0], static_cast<int32_T>(dim));
          v_outdatedOnGpu = true;
        }
        npages = cpu_v->size[1];
        i = b_cpu_y->size[0] * b_cpu_y->size[1];
        b_cpu_y->size[0] = 1;
        b_cpu_y->size[1] = cpu_v->size[1];
        emxEnsureCapacity_real_T(b_cpu_y, i, &o_emlrtRTEI);
        mwGetLaunchParameters1D(computeNumIters(npages - 1), &grid, &block,
                                2147483647U);
        gpuEmxEnsureCapacity_real_T(cpu_v, &gpu_v, !v_outdatedOnGpu);
        gpuEmxEnsureCapacity_real_T(b_cpu_y, &d_gpu_y, true);
        if (v_outdatedOnGpu) {
          gpuEmxMemcpyCpuToGpu_real_T(&gpu_v, cpu_v);
        }
        v_outdatedOnGpu = false;
        validLaunchParams = mwValidateLaunchParameters(grid, block);
        if (validLaunchParams) {
          n_parrot_optimizer_animated_ker<<<grid, block>>>(gpu_v, npages - 1,
                                                           d_gpu_y);
        }
        hipDeviceSynchronize();
        checkCudaError(hipGetLastError(), __FILE__, __LINE__);
        i = d_cpu_y->size[0] * d_cpu_y->size[1];
        d_cpu_y->size[0] = 1;
        d_cpu_y->size[1] = b_cpu_y->size[1];
        emxEnsureCapacity_real_T(d_cpu_y, i, &p_emlrtRTEI);
        npages = b_cpu_y->size[1];
        mwGetLaunchParameters1D(computeNumIters(npages - 1), &grid, &block,
                                2147483647U);
        gpuEmxEnsureCapacity_real_T(d_cpu_y, &e_gpu_y, true);
        validLaunchParams = mwValidateLaunchParameters(grid, block);
        if (validLaunchParams) {
          o_parrot_optimizer_animated_ker<<<grid, block>>>(d_gpu_y, npages - 1,
                                                           e_gpu_y);
        }
        hipDeviceSynchronize();
        checkCudaError(hipGetLastError(), __FILE__, __LINE__);
        i = cpu_dv5->size[0] * cpu_dv5->size[1];
        cpu_dv5->size[0] = 1;
        cpu_dv5->size[1] = static_cast<int32_T>(dim);
        emxEnsureCapacity_real_T(cpu_dv5, i, &emlrtRTEI);
        if (static_cast<int32_T>(dim) != 0) {
          emlrtRandu(&cpu_dv5->data[0], static_cast<int32_T>(dim));
          dv5_outdatedOnGpu = true;
        }
        if (b_cpu_r->size[1] == 1) {
          vlen = d_cpu_y->size[1];
        } else {
          vlen = b_cpu_r->size[1];
        }
        if (cpu_X_best->size[1] == 1) {
          if (b_cpu_r->size[1] == 1) {
            i = d_cpu_y->size[1];
          } else {
            i = b_cpu_r->size[1];
          }
        } else {
          i = cpu_X_best->size[1];
        }
        if (cpu_X->size[1] == 1) {
          if (cpu_X_best->size[1] == 1) {
            if (b_cpu_r->size[1] == 1) {
              npages = d_cpu_y->size[1];
            } else {
              npages = b_cpu_r->size[1];
            }
          } else {
            npages = cpu_X_best->size[1];
          }
        } else {
          npages = cpu_X->size[1];
        }
        if ((b_cpu_r->size[1] == d_cpu_y->size[1]) &&
            (cpu_X_best->size[1] == vlen) && (cpu_X->size[1] == i) &&
            (npages == cpu_dv5->size[1])) {
          i = cpu_X_new->size[0] * cpu_X_new->size[1];
          cpu_X_new->size[0] = 1;
          cpu_X_new->size[1] = cpu_X->size[1];
          emxEnsureCapacity_real_T(cpu_X_new, i, &r_emlrtRTEI);
          vlen = cpu_X->size[1] - 1;
          mwGetLaunchParameters1D(computeNumIters(vlen), &grid, &block,
                                  2147483647U);
          gpuEmxEnsureCapacity_real_T(cpu_dv5, &gpu_dv5, !dv5_outdatedOnGpu);
          gpuEmxEnsureCapacity_real_T(b_cpu_r, &c_gpu_r, !b_r_outdatedOnGpu);
          if (X_best_needsGpuEnsureCapacity) {
            gpuEmxEnsureCapacity_real_T(cpu_X_best, &gpu_X_best, true);
          }
          X_best_needsGpuEnsureCapacity = false;
          if (X_needsGpuEnsureCapacity) {
            gpuEmxEnsureCapacity_real_T(cpu_X, &gpu_X, true);
          }
          X_needsGpuEnsureCapacity = false;
          gpuEmxEnsureCapacity_real_T(cpu_X_new, &gpu_X_new, true);
          X_new_needsGpuEnsureCapacity = false;
          if (dv5_outdatedOnGpu) {
            gpuEmxMemcpyCpuToGpu_real_T(&gpu_dv5, cpu_dv5);
          }
          dv5_outdatedOnGpu = false;
          if (b_r_outdatedOnGpu) {
            gpuEmxMemcpyCpuToGpu_real_T(&c_gpu_r, b_cpu_r);
          }
          b_r_outdatedOnGpu = false;
          validLaunchParams = mwValidateLaunchParameters(grid, block);
          if (validLaunchParams) {
            p_parrot_optimizer_animated_ker<<<grid, block>>>(
                gpu_dv5, e_gpu_y, c_gpu_r, gpu_X_best, gpu_X, b_i, vlen,
                gpu_X_new, cpu_X->size[0U]);
          }
          hipDeviceSynchronize();
          checkCudaError(hipGetLastError(), __FILE__, __LINE__);
          p = false;
        } else {
          if (X_outdatedOnCpu) {
            gpuEmxMemcpyGpuToCpu_real_T(cpu_X, &gpu_X);
          }
          X_outdatedOnCpu = false;
          if (X_best_outdatedOnCpu) {
            gpuEmxMemcpyGpuToCpu_real_T(cpu_X_best, &gpu_X_best);
          }
          X_best_outdatedOnCpu = false;
          gpuEmxMemcpyGpuToCpu_real_T(d_cpu_y, &e_gpu_y);
          binary_expand_op_1(cpu_X_new, cpu_X, b_i, cpu_X_best, b_cpu_r,
                             d_cpu_y, cpu_dv5);
          p = true;
          X_best_needsGpuEnsureCapacity = true;
          X_needsGpuEnsureCapacity = true;
          X_new_needsGpuEnsureCapacity = true;
        }
        //  行为3：交流行为
      } else if (St + 1.0 == 3.0) {
        emlrtRandu(&St, 1);
        if (St <= 0.5) {
          emlrtRandu(&St, 1);
          t2 = 0.2 * St * (1.0 - (static_cast<real_T>(t) + 1.0) / Max_iter);
          if (cpu_X->size[1] == cpu_X_mean->size[1]) {
            i = cpu_X_new->size[0] * cpu_X_new->size[1];
            cpu_X_new->size[0] = 1;
            cpu_X_new->size[1] = cpu_X->size[1];
            emxEnsureCapacity_real_T(cpu_X_new, i, &n_emlrtRTEI);
            vlen = cpu_X->size[1] - 1;
            mwGetLaunchParameters1D(computeNumIters(vlen), &grid, &block,
                                    2147483647U);
            if (X_mean_needsGpuEnsureCapacity) {
              gpuEmxEnsureCapacity_real_T(cpu_X_mean, &gpu_X_mean, true);
            }
            X_mean_needsGpuEnsureCapacity = false;
            if (X_needsGpuEnsureCapacity) {
              gpuEmxEnsureCapacity_real_T(cpu_X, &gpu_X, true);
            }
            X_needsGpuEnsureCapacity = false;
            gpuEmxEnsureCapacity_real_T(cpu_X_new, &gpu_X_new, true);
            X_new_needsGpuEnsureCapacity = false;
            validLaunchParams = mwValidateLaunchParameters(grid, block);
            if (validLaunchParams) {
              q_parrot_optimizer_animated_ker<<<grid, block>>>(
                  gpu_X_mean, gpu_X, b_i, t2, vlen, gpu_X_new, cpu_X->size[0U]);
            }
            hipDeviceSynchronize();
            checkCudaError(hipGetLastError(), __FILE__, __LINE__);
            p = false;
          } else {
            if (X_outdatedOnCpu) {
              gpuEmxMemcpyGpuToCpu_real_T(cpu_X, &gpu_X);
            }
            X_outdatedOnCpu = false;
            if (X_mean_outdatedOnCpu) {
              gpuEmxMemcpyGpuToCpu_real_T(cpu_X_mean, &gpu_X_mean);
            }
            X_mean_outdatedOnCpu = false;
            binary_expand_op_2(cpu_X_new, t2, cpu_X, b_i, cpu_X_mean);
            p = true;
            X_mean_needsGpuEnsureCapacity = true;
            X_needsGpuEnsureCapacity = true;
            X_new_needsGpuEnsureCapacity = true;
          }
        } else {
          r_outdatedOnGpu = false;
          i = d_cpu_r->size[0] * d_cpu_r->size[1];
          d_cpu_r->size[0] = 1;
          d_cpu_r->size[1] = static_cast<int32_T>(dim);
          emxEnsureCapacity_real_T(d_cpu_r, i, &emlrtRTEI);
          if (static_cast<int32_T>(dim) != 0) {
            emlrtRandu(&d_cpu_r->data[0], static_cast<int32_T>(dim));
            r_outdatedOnGpu = true;
          }
          p = false;
          i = e_cpu_r->size[0] * e_cpu_r->size[1];
          e_cpu_r->size[0] = 1;
          e_cpu_r->size[1] = static_cast<int32_T>(dim);
          emxEnsureCapacity_real_T(e_cpu_r, i, &emlrtRTEI);
          if (static_cast<int32_T>(dim) != 0) {
            emlrtRandu(&e_cpu_r->data[0], static_cast<int32_T>(dim));
            p = true;
          }
          i = cpu_dv4->size[0] * cpu_dv4->size[1];
          cpu_dv4->size[0] = 1;
          cpu_dv4->size[1] = e_cpu_r->size[1];
          emxEnsureCapacity_real_T(cpu_dv4, i, &q_emlrtRTEI);
          vlen = e_cpu_r->size[1] - 1;
          mwGetLaunchParameters1D(computeNumIters(vlen), &grid, &block,
                                  2147483647U);
          gpuEmxEnsureCapacity_real_T(e_cpu_r, &d_gpu_r, !p);
          gpuEmxEnsureCapacity_real_T(cpu_dv4, &gpu_dv4, true);
          if (p) {
            gpuEmxMemcpyCpuToGpu_real_T(&d_gpu_r, e_cpu_r);
          }
          validLaunchParams = mwValidateLaunchParameters(grid, block);
          if (validLaunchParams) {
            r_parrot_optimizer_animated_ker<<<grid, block>>>(
                Max_iter, d_gpu_r,
                static_cast<int32_T>(-(static_cast<real_T>(t) + 1.0)), vlen,
                gpu_dv4);
          }
          hipDeviceSynchronize();
          checkCudaError(hipGetLastError(), __FILE__, __LINE__);
          npages = cpu_dv4->size[1];
          mwGetLaunchParameters1D(computeNumIters(npages - 1), &grid, &block,
                                  2147483647U);
          validLaunchParams = mwValidateLaunchParameters(grid, block);
          if (validLaunchParams) {
            s_parrot_optimizer_animated_ker<<<grid, block>>>(npages - 1,
                                                             gpu_dv4);
          }
          hipDeviceSynchronize();
          checkCudaError(hipGetLastError(), __FILE__, __LINE__);
          if (d_cpu_r->size[1] == 1) {
            vlen = cpu_dv4->size[1];
          } else {
            vlen = d_cpu_r->size[1];
          }
          if ((d_cpu_r->size[1] == cpu_dv4->size[1]) &&
              (cpu_X->size[1] == vlen)) {
            i = cpu_X_new->size[0] * cpu_X_new->size[1];
            cpu_X_new->size[0] = 1;
            cpu_X_new->size[1] = cpu_X->size[1];
            emxEnsureCapacity_real_T(cpu_X_new, i, &t_emlrtRTEI);
            vlen = cpu_X->size[1] - 1;
            mwGetLaunchParameters1D(computeNumIters(vlen), &grid, &block,
                                    2147483647U);
            gpuEmxEnsureCapacity_real_T(d_cpu_r, &e_gpu_r, !r_outdatedOnGpu);
            if (X_needsGpuEnsureCapacity) {
              gpuEmxEnsureCapacity_real_T(cpu_X, &gpu_X, true);
            }
            X_needsGpuEnsureCapacity = false;
            gpuEmxEnsureCapacity_real_T(cpu_X_new, &gpu_X_new, true);
            X_new_needsGpuEnsureCapacity = false;
            if (r_outdatedOnGpu) {
              gpuEmxMemcpyCpuToGpu_real_T(&e_gpu_r, d_cpu_r);
            }
            validLaunchParams = mwValidateLaunchParameters(grid, block);
            if (validLaunchParams) {
              t_parrot_optimizer_animated_ker<<<grid, block>>>(
                  gpu_dv4, e_gpu_r, gpu_X, b_i, vlen, gpu_X_new,
                  cpu_X->size[0U]);
            }
            hipDeviceSynchronize();
            checkCudaError(hipGetLastError(), __FILE__, __LINE__);
            p = false;
          } else {
            if (X_outdatedOnCpu) {
              gpuEmxMemcpyGpuToCpu_real_T(cpu_X, &gpu_X);
            }
            X_outdatedOnCpu = false;
            gpuEmxMemcpyGpuToCpu_real_T(cpu_dv4, &gpu_dv4);
            binary_expand_op_3(cpu_X_new, cpu_X, b_i, d_cpu_r, cpu_dv4);
            p = true;
            X_needsGpuEnsureCapacity = true;
            X_new_needsGpuEnsureCapacity = true;
          }
          //  添加随机扰动向量
        }
        //  行为4：恐惧行为
      } else {
        emlrtRandu(&St, 1);
        t2 = St * std::cos(2.5132741228718345 * (static_cast<real_T>(t) + 1.0) /
                           Max_iter);
        emlrtRandu(&St, 1);
        St = std::cos(St * 3.1415926535897931) *
             rt_powd_snf((static_cast<real_T>(t) + 1.0) / Max_iter,
                         2.0 / Max_iter);
        if (cpu_X_best->size[1] == 1) {
          vlen = cpu_X->size[1];
        } else {
          vlen = cpu_X_best->size[1];
        }
        if (cpu_X->size[1] == 1) {
          if (cpu_X_best->size[1] == 1) {
            i = cpu_X->size[1];
          } else {
            i = cpu_X_best->size[1];
          }
        } else {
          i = cpu_X->size[1];
        }
        if (cpu_X->size[1] == 1) {
          npages = cpu_X_best->size[1];
        } else {
          npages = cpu_X->size[1];
        }
        if ((cpu_X_best->size[1] == cpu_X->size[1]) &&
            (cpu_X->size[1] == vlen) &&
            (cpu_X->size[1] == cpu_X_best->size[1]) && (i == npages)) {
          i = cpu_X_new->size[0] * cpu_X_new->size[1];
          cpu_X_new->size[0] = 1;
          cpu_X_new->size[1] = cpu_X->size[1];
          emxEnsureCapacity_real_T(cpu_X_new, i, &m_emlrtRTEI);
          vlen = cpu_X->size[1] - 1;
          mwGetLaunchParameters1D(computeNumIters(vlen), &grid, &block,
                                  2147483647U);
          if (X_needsGpuEnsureCapacity) {
            gpuEmxEnsureCapacity_real_T(cpu_X, &gpu_X, true);
          }
          X_needsGpuEnsureCapacity = false;
          if (X_best_needsGpuEnsureCapacity) {
            gpuEmxEnsureCapacity_real_T(cpu_X_best, &gpu_X_best, true);
          }
          X_best_needsGpuEnsureCapacity = false;
          gpuEmxEnsureCapacity_real_T(cpu_X_new, &gpu_X_new, true);
          X_new_needsGpuEnsureCapacity = false;
          validLaunchParams = mwValidateLaunchParameters(grid, block);
          if (validLaunchParams) {
            u_parrot_optimizer_animated_ker<<<grid, block>>>(
                St, t2, gpu_X, b_i, gpu_X_best, vlen, gpu_X_new,
                cpu_X->size[0U]);
          }
          hipDeviceSynchronize();
          checkCudaError(hipGetLastError(), __FILE__, __LINE__);
          p = false;
        } else {
          if (X_outdatedOnCpu) {
            gpuEmxMemcpyGpuToCpu_real_T(cpu_X, &gpu_X);
          }
          X_outdatedOnCpu = false;
          if (X_best_outdatedOnCpu) {
            gpuEmxMemcpyGpuToCpu_real_T(cpu_X_best, &gpu_X_best);
          }
          X_best_outdatedOnCpu = false;
          binary_expand_op_4(cpu_X_new, cpu_X, b_i, t2, cpu_X_best, St);
          p = true;
          X_best_needsGpuEnsureCapacity = true;
          X_needsGpuEnsureCapacity = true;
          X_new_needsGpuEnsureCapacity = true;
        }
      }
      //  边界约束
      i = cpu_maxval->size[0] * cpu_maxval->size[1];
      cpu_maxval->size[0] = 1;
      cpu_maxval->size[1] = cpu_X_new->size[1];
      emxEnsureCapacity_real_T(cpu_maxval, i, &p_emlrtRTEI);
      npages = cpu_X_new->size[1];
      mwGetLaunchParameters1D(computeNumIters(npages - 1), &grid, &block,
                              2147483647U);
      if (X_new_needsGpuEnsureCapacity) {
        gpuEmxEnsureCapacity_real_T(cpu_X_new, &gpu_X_new, !p);
      }
      gpuEmxEnsureCapacity_real_T(cpu_maxval, &gpu_maxval, true);
      if (p) {
        gpuEmxMemcpyCpuToGpu_real_T(&gpu_X_new, cpu_X_new);
      }
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        k_parrot_optimizer_animated_ker<<<grid, block>>>(
            lb, gpu_X_new, npages - 1, gpu_maxval);
      }
      hipDeviceSynchronize();
      checkCudaError(hipGetLastError(), __FILE__, __LINE__);
      i = cpu_X_new->size[0] * cpu_X_new->size[1];
      cpu_X_new->size[0] = 1;
      cpu_X_new->size[1] = cpu_maxval->size[1];
      emxEnsureCapacity_real_T(cpu_X_new, i, &p_emlrtRTEI);
      npages = cpu_maxval->size[1];
      mwGetLaunchParameters1D(computeNumIters(npages - 1), &grid, &block,
                              2147483647U);
      gpuEmxEnsureCapacity_real_T(cpu_X_new, &gpu_X_new, true);
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        l_parrot_optimizer_animated_ker<<<grid, block>>>(ub, gpu_maxval,
                                                         npages - 1, gpu_X_new);
      }
      hipDeviceSynchronize();
      checkCudaError(hipGetLastError(), __FILE__, __LINE__);
      //  贪婪选择
      //  输入：x为N×2矩阵，每行代表一个个体（二维）
      //  输出：N×1向量，每个个体的适应度值
      St = 0.0;
      t2 = 0.0;
      gpuEmxMemcpyGpuToCpu_real_T(cpu_X_new, &gpu_X_new);
      d2 = cpu_X_new->data[0];
      d3 = cpu_X_new->data[1];
      for (npages = 0; npages < 5; npages++) {
        St += (static_cast<real_T>(npages) + 1.0) *
              std::cos(((static_cast<real_T>(npages) + 1.0) + 1.0) * d2 +
                       (static_cast<real_T>(npages) + 1.0));
        t2 += (static_cast<real_T>(npages) + 1.0) *
              std::cos(((static_cast<real_T>(npages) + 1.0) + 1.0) * d3 +
                       (static_cast<real_T>(npages) + 1.0));
      }
      St *= t2;
      if (St < cpu_fitness->data[b_i]) {
        npages = cpu_X->size[1];
        mwGetLaunchParameters1D(computeNumIters(npages - 1), &grid, &block,
                                2147483647U);
        gpuEmxEnsureCapacity_real_T(cpu_X_new, &gpu_X_new, true);
        if (X_needsGpuEnsureCapacity) {
          gpuEmxEnsureCapacity_real_T(cpu_X, &gpu_X, true);
        }
        X_needsGpuEnsureCapacity = false;
        validLaunchParams = mwValidateLaunchParameters(grid, block);
        if (validLaunchParams) {
          m_parrot_optimizer_animated_ker<<<grid, block>>>(
              gpu_X_new, b_i, npages - 1, gpu_X, cpu_X->size[0U]);
        }
        hipDeviceSynchronize();
        checkCudaError(hipGetLastError(), __FILE__, __LINE__);
        X_outdatedOnCpu = true;
        cpu_fitness->data[b_i] = St;
      }
    }
    //  更新全局最优
    npages = 0;
    vlen = cpu_fitness->size[0];
    St = cpu_fitness->data[0];
    for (b_i = 0; b_i <= vlen - 2; b_i++) {
      if (std::isnan(cpu_fitness->data[b_i + 1])) {
        p = false;
      } else if (std::isnan(St)) {
        p = true;
      } else {
        p = (St > cpu_fitness->data[b_i + 1]);
      }
      if (p) {
        St = cpu_fitness->data[b_i + 1];
        npages = b_i + 1;
      }
    }
    if (St < ex) {
      ex = St;
      i = cpu_X_best->size[0] * cpu_X_best->size[1];
      cpu_X_best->size[0] = 1;
      cpu_X_best->size[1] = cpu_X->size[1];
      emxEnsureCapacity_real_T(cpu_X_best, i, &l_emlrtRTEI);
      vlen = cpu_X->size[1] - 1;
      mwGetLaunchParameters1D(computeNumIters(vlen), &grid, &block,
                              2147483647U);
      if (X_needsGpuEnsureCapacity) {
        gpuEmxEnsureCapacity_real_T(cpu_X, &gpu_X, true);
      }
      X_needsGpuEnsureCapacity = false;
      gpuEmxEnsureCapacity_real_T(cpu_X_best, &gpu_X_best, true);
      X_best_needsGpuEnsureCapacity = false;
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        v_parrot_optimizer_animated_ker<<<grid, block>>>(
            gpu_X, npages, vlen, gpu_X_best, cpu_X->size[0U]);
      }
      hipDeviceSynchronize();
      checkCudaError(hipGetLastError(), __FILE__, __LINE__);
      X_best_outdatedOnCpu = true;
    }
    Convergence_curve->data[t] = ex;
  }
  emxFree_real_T(&cpu_dv5);
  emxFree_real_T(&e_cpu_y);
  emxFree_real_T(&d_cpu_y);
  emxFree_real_T(&cpu_maxval);
  emxFree_real_T(&cpu_dv4);
  emxFree_real_T(&c_cpu_y);
  emxFree_real_T(&b_cpu_y);
  emxFree_real_T(&e_cpu_r);
  emxFree_real_T(&cpu_X_new);
  emxFree_real_T(&cpu_v);
  emxFree_real_T(&d_cpu_r);
  emxFree_real_T(&c_cpu_r);
  emxFree_real_T(&b_cpu_r);
  emxFree_real_T(&cpu_X_mean);
  emxFree_real_T(&cpu_y);
  emxFree_real_T(&cpu_fitness);
  emxFree_real_T(&cpu_X);
  *f_best = ex;
  emlrtHeapReferenceStackLeaveFcnR2012b(emlrtRootTLSGlobal);
  if (X_best_outdatedOnCpu) {
    gpuEmxMemcpyGpuToCpu_real_T(cpu_X_best, &gpu_X_best);
  }
  gpuEmxFree_real_T(&gpu_X_best);
  gpuEmxFree_real_T(&gpu_r);
  gpuEmxFree_real_T(&gpu_X);
  gpuEmxFree_real_T(&gpu_fitness);
  gpuEmxFree_real_T(&gpu_y);
  gpuEmxFree_real_T(&gpu_X_mean);
  gpuEmxFree_real_T(&c_gpu_r);
  gpuEmxFree_real_T(&b_gpu_r);
  gpuEmxFree_real_T(&e_gpu_r);
  gpuEmxFree_real_T(&gpu_v);
  gpuEmxFree_real_T(&gpu_X_new);
  gpuEmxFree_real_T(&d_gpu_r);
  gpuEmxFree_real_T(&d_gpu_y);
  gpuEmxFree_real_T(&b_gpu_y);
  gpuEmxFree_real_T(&gpu_dv4);
  gpuEmxFree_real_T(&gpu_maxval);
  gpuEmxFree_real_T(&e_gpu_y);
  gpuEmxFree_real_T(&c_gpu_y);
  gpuEmxFree_real_T(&gpu_dv5);
}

// End of code generation (parrot_optimizer_animated.cu)
